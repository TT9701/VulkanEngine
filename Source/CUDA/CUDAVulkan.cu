#include "hip/hip_runtime.h"
#define VULKAN_HPP_NO_SPACESHIP_OPERATOR
#define VULKAN_HPP_NO_TO_STRING
#include "CUDAVulkan.h"

#include <vulkan/vulkan_win32.h>
#include <cassert>
#include <cmath>
#include <stdexcept>

#include "../Core/MeshType.hpp"

namespace {

__global__ void SimpleAdd(void* data, float time) {
    auto vertices = static_cast<Vertex*>(data);

    vertices[0].position = {0.0f, 0.0f, 0.0f};
    vertices[1].position = {1.0f, 0.0f, 0.0f};
    vertices[2].position = {0.5f, 0.5f * ::std::sin(time / 1000.0f), 0.0f};

    vertices[0].color = {1.0f, 0.0f, 0.0f, 1.0f};
    vertices[1].color = {0.0f, 1.0f, 0.0f, 1.0f};
    vertices[2].color = {0.0f, 0.0f, 1.0f, 1.0f};

    vertices[0].uvX = 1.0f;
    vertices[1].uvX = 0.0f;
    vertices[2].uvX = 0.5f;

    vertices[0].uvY = 0.0f;
    vertices[1].uvY = 0.0f;
    vertices[2].uvY = 1.0f;
}

hipExternalMemory_t GetCUDAExternalMemory(VkDevice device, VkDeviceMemory vkDeviceMemory, size_t allocByteSize) {
    PFN_vkGetMemoryWin32HandleKHR fpGetMemoryWin32HandleKHR =
        (PFN_vkGetMemoryWin32HandleKHR)vkGetDeviceProcAddr(
            device, "vkGetMemoryWin32HandleKHR");

    if (!fpGetMemoryWin32HandleKHR) {
        throw std::runtime_error(
            "Failed to retrieve vkGetMemoryWin32HandleKHR!");
    }

    VkMemoryGetWin32HandleInfoKHR memoryWin32HandleInfo {};
    memoryWin32HandleInfo.sType =
        VK_STRUCTURE_TYPE_MEMORY_GET_WIN32_HANDLE_INFO_KHR;
    memoryWin32HandleInfo.handleType =
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT;
    memoryWin32HandleInfo.memory = vkDeviceMemory;

    HANDLE handle {};

    if (fpGetMemoryWin32HandleKHR(device, &memoryWin32HandleInfo, &handle) !=
        VK_SUCCESS) {
        throw std::runtime_error("Failed to retrieve handle for buffer!");
    }

    hipExternalMemoryHandleDesc desc {};

    memset(&desc, 0, sizeof(desc));

    desc.type                = hipExternalMemoryHandleTypeOpaqueWin32;
    desc.handle.win32.handle = handle;
    desc.size                = allocByteSize;
    desc.flags |= hipExternalMemoryDedicated;

    hipExternalMemory_t extMem {};
    hipImportExternalMemory(&extMem, &desc);

    CloseHandle(handle);

    return extMem;
}

}  // namespace

namespace CUDA {

int GetVulkanCUDABindDeviceID(vk::PhysicalDevice vkPhysicalDevice) {
    vk::PhysicalDeviceIDProperties vkPhysicalDeviceIDProperties {};

    vk::PhysicalDeviceProperties2 vkPhysicalDeviceProperties2 {};
    vkPhysicalDeviceProperties2.pNext = &vkPhysicalDeviceIDProperties;
    vkPhysicalDevice.getProperties2(&vkPhysicalDeviceProperties2);

    int cudaDeviceCount;
    hipGetDeviceCount(&cudaDeviceCount);

    for (int cudaDevice = 0; cudaDevice < cudaDeviceCount; cudaDevice++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, cudaDevice);
        if (!memcmp(&deviceProp.uuid, vkPhysicalDeviceIDProperties.deviceUUID,
                    VK_UUID_SIZE)) {
            return cudaDevice;
        }
    }
    return hipInvalidDeviceId;
}

void VulkanExternalBuffer::CreateExternalBuffer(
    vk::Device device, VmaAllocator allocator, size_t allocByteSize,
    vk::BufferUsageFlags usage, VmaAllocationCreateFlags flags, VmaPool pool) {
    vk::ExternalMemoryBufferCreateInfo externalbuffer {};
    externalbuffer.setHandleTypes(
        vk::ExternalMemoryHandleTypeFlagBits::eOpaqueWin32);

    vk::BufferCreateInfo bufferInfo {};
    bufferInfo.setSize(allocByteSize).setUsage(usage).setPNext(&externalbuffer);

    VmaAllocationCreateInfo vmaAllocInfo {};
    vmaAllocInfo.usage = VMA_MEMORY_USAGE_AUTO;
    vmaAllocInfo.flags = flags;
    vmaAllocInfo.pool  = pool;

    vmaCreateBuffer(allocator, (VkBufferCreateInfo*)&bufferInfo, &vmaAllocInfo,
                    (VkBuffer*)&mBuffer, &mAllocation, &mInfo);

    mExternalMemory =
        GetCUDAExternalMemory(device, mInfo.deviceMemory, allocByteSize);

    mAllocator = allocator;
}

void VulkanExternalSemaphore::CreateExternalSemaphore(vk::Device device) {
    vk::SemaphoreCreateInfo semaphoreInfo {};

    vk::ExportSemaphoreCreateInfoKHR exportSemaphoreCreateInfo {};
    exportSemaphoreCreateInfo.setHandleTypes(
        vk::ExternalSemaphoreHandleTypeFlagBits::eOpaqueWin32);
    semaphoreInfo.setPNext(&exportSemaphoreCreateInfo);

    mSemaphore = device.createSemaphore(semaphoreInfo);

    VkSemaphoreGetWin32HandleInfoKHR semaphoreGetWin32HandleInfoKHR = {};
    semaphoreGetWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_SEMAPHORE_GET_WIN32_HANDLE_INFO_KHR;
    semaphoreGetWin32HandleInfoKHR.pNext     = nullptr;
    semaphoreGetWin32HandleInfoKHR.semaphore = mSemaphore;
    semaphoreGetWin32HandleInfoKHR.handleType =
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT;

    PFN_vkGetSemaphoreWin32HandleKHR fpGetSemaphoreWin32HandleKHR;
    fpGetSemaphoreWin32HandleKHR =
        (PFN_vkGetSemaphoreWin32HandleKHR)vkGetDeviceProcAddr(
            device, "vkGetSemaphoreWin32HandleKHR");

    if (!fpGetSemaphoreWin32HandleKHR) {
        throw std::runtime_error(
            "Failed to retrieve vkGetSemaphoreWin32HandleKHR!");
    }

    HANDLE handle;

    if (fpGetSemaphoreWin32HandleKHR(device, &semaphoreGetWin32HandleInfoKHR,
                                     &handle) != VK_SUCCESS) {
        throw std::runtime_error("Failed to retrieve handle for Semaphore!");
    }

    hipExternalSemaphoreHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));

    desc.type                = hipExternalSemaphoreHandleTypeOpaqueWin32;
    desc.handle.win32.handle = handle;

    hipImportExternalSemaphore(&mExternalSemaphore, &desc);

    CloseHandle(handle);
}

void VulkanExternalSemaphore::InsertWaitToStreamAsync(hipStream_t cudaStream) {
    hipExternalSemaphoreWaitParams params = {};

    memset(&params, 0, sizeof(params));

    hipWaitExternalSemaphoresAsync(&mExternalSemaphore, &params, 1, cudaStream);
}

void VulkanExternalSemaphore::InsertSignalToStreamAsync(
    hipStream_t cudaStream) {
    hipExternalSemaphoreSignalParams params = {};

    memset(&params, 0, sizeof(params));

    hipSignalExternalSemaphoresAsync(&mExternalSemaphore, &params, 1, cudaStream);
}

hipMipmappedArray_t MapMipmappedArrayOntoExternalMemory(
    hipExternalMemory_t extMem, unsigned long long offset,
    hipChannelFormatDesc* formatDesc, hipExtent* extent, unsigned int flags,
    unsigned int numLevels) {
    hipMipmappedArray_t                 mipmap = NULL;
    cudaExternalMemoryMipmappedArrayDesc desc   = {};

    memset(&desc, 0, sizeof(desc));

    desc.offset     = offset;
    desc.formatDesc = *formatDesc;
    desc.extent     = *extent;
    desc.flags      = flags;
    desc.numLevels  = numLevels;

    // Note: 'mipmap' must eventually be freed using hipFreeMipmappedArray()
    cudaExternalMemoryGetMappedMipmappedArray(&mipmap, extMem, &desc);

    return mipmap;
}

hipChannelFormatDesc GetCudaChannelFormatDescForVulkanFormat(
    vk::Format format) {
    hipChannelFormatDesc d;

    memset(&d, 0, sizeof(d));

    switch (format) {
        case vk::Format::eR8Uint:
            d.x = 8;
            d.y = 0;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR8Sint:
            d.x = 8;
            d.y = 0;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR8G8Uint:
            d.x = 8;
            d.y = 8;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR8G8Sint:
            d.x = 8;
            d.y = 8;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR8G8B8A8Uint:
            d.x = 8;
            d.y = 8;
            d.z = 8;
            d.w = 8;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR8G8B8A8Sint:
            d.x = 8;
            d.y = 8;
            d.z = 8;
            d.w = 8;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR16Uint:
            d.x = 16;
            d.y = 0;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR16Sint:
            d.x = 16;
            d.y = 0;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR16G16Uint:
            d.x = 16;
            d.y = 16;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR16G16Sint:
            d.x = 16;
            d.y = 16;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR16G16B16A16Uint:
            d.x = 16;
            d.y = 16;
            d.z = 16;
            d.w = 16;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR16G16B16A16Sint:
            d.x = 16;
            d.y = 16;
            d.z = 16;
            d.w = 16;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR32Uint:
            d.x = 32;
            d.y = 0;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR32Sint:
            d.x = 32;
            d.y = 0;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR32Sfloat:
            d.x = 32;
            d.y = 0;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindFloat;
            break;
        case vk::Format::eR32G32Uint:
            d.x = 32;
            d.y = 32;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR32G32Sint:
            d.x = 32;
            d.y = 32;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR32G32Sfloat:
            d.x = 32;
            d.y = 32;
            d.z = 0;
            d.w = 0;
            d.f = hipChannelFormatKindFloat;
            break;
        case vk::Format::eR32G32B32A32Uint:
            d.x = 32;
            d.y = 32;
            d.z = 32;
            d.w = 32;
            d.f = hipChannelFormatKindUnsigned;
            break;
        case vk::Format::eR32G32B32A32Sint:
            d.x = 32;
            d.y = 32;
            d.z = 32;
            d.w = 32;
            d.f = hipChannelFormatKindSigned;
            break;
        case vk::Format::eR32G32B32A32Sfloat:
            d.x = 32;
            d.y = 32;
            d.z = 32;
            d.w = 32;
            d.f = hipChannelFormatKindFloat;
            break;
        default:
            assert(0);
    }
    return d;
}

hipExtent GetCudaExtentForVulkanExtent(vk::Extent3D      vkExt,
                                        uint32_t          arrayLayers,
                                        vk::ImageViewType vkImageViewType) {
    hipExtent e = {0, 0, 0};

    switch (vkImageViewType) {
        case vk::ImageViewType::e1D:
            e.width  = vkExt.width;
            e.height = 0;
            e.depth  = 0;
            break;
        case vk::ImageViewType::e2D:
            e.width  = vkExt.width;
            e.height = vkExt.height;
            e.depth  = 0;
            break;
        case vk::ImageViewType::e3D:
            e.width  = vkExt.width;
            e.height = vkExt.height;
            e.depth  = vkExt.depth;
            break;
        case vk::ImageViewType::eCube:
            e.width  = vkExt.width;
            e.height = vkExt.height;
            e.depth  = arrayLayers;
            break;
        case vk::ImageViewType::e1DArray:
            e.width  = vkExt.width;
            e.height = 0;
            e.depth  = arrayLayers;
            break;
        case vk::ImageViewType::e2DArray:
            e.width  = vkExt.width;
            e.height = vkExt.height;
            e.depth  = arrayLayers;
            break;
        case vk::ImageViewType::eCubeArray:
            e.width  = vkExt.width;
            e.height = vkExt.height;
            e.depth  = arrayLayers;
            break;
        default:
            assert(0);
    }

    return e;
}

unsigned int GetCudaMipmappedArrayFlagsForVulkanImage(
    vk::ImageViewType vkImageViewType, vk::ImageUsageFlags vkImageUsageFlags,
    bool allowSurfaceLoadStore) {
    unsigned int flags = 0;

    switch (vkImageViewType) {
        case vk::ImageViewType::eCube:
            flags |= hipArrayCubemap;
            break;
        case vk::ImageViewType::eCubeArray:
            flags |= hipArrayCubemap | hipArrayLayered;
            break;
        case vk::ImageViewType::e1DArray:
            flags |= hipArrayLayered;
            break;
        case vk::ImageViewType::e2DArray:
            flags |= hipArrayLayered;
            break;
        default:
            break;
    }

    if (vkImageUsageFlags & vk::ImageUsageFlagBits::eColorAttachment) {
        flags |= cudaArrayColorAttachment;
    }

    if (allowSurfaceLoadStore) {
        flags |= hipArraySurfaceLoadStore;
    }
    return flags;
}

void SimPoint(void* data, float time) {
    SimpleAdd<<<1, 1>>>(data, time);
}

}  // namespace CUDA
