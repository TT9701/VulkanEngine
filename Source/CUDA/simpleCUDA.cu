#include "simpleCUDA.h"

float* MatAdd(float* a, float* b, int length) {
    int device = 0;
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    int    threadMaxSize = devProp.maxThreadsPerBlock;
    int    blockSize     = (length + threadMaxSize - 1) / threadMaxSize;
    dim3   thread(threadMaxSize);
    dim3   block(blockSize);
    int    size = length * sizeof(float);
    float* sum  = (float*)malloc(size);
    float *sumGPU, *aGPU, *bGPU;
    hipMalloc((void**)&sumGPU, size);
    hipMalloc((void**)&aGPU, size);
    hipMalloc((void**)&bGPU, size);
    hipMemcpy((void*)aGPU, (void*)a, size, hipMemcpyHostToDevice);
    hipMemcpy((void*)bGPU, (void*)b, size, hipMemcpyHostToDevice);
    CudaAdd<float><<<block, thread>>>(aGPU, bGPU, sumGPU);
    //hipDeviceSynchronize();
    hipMemcpy(sum, sumGPU, size, hipMemcpyDeviceToHost);
    hipFree(sumGPU);
    hipFree(aGPU);
    hipFree(bGPU);
    return sum;
}