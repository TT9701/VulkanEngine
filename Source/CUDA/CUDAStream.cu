#include "CUDAStream.h"

namespace CUDA {

void CUDAStream::Synchronize() const {
    HANDLE_ERROR(hipStreamSynchronize(mStream));
}

void CUDAStream::EventSynchronize(hipEvent_t cuEvent) const {
    HANDLE_ERROR(hipEventSynchronize(cuEvent));
}

void CUDAStream::WaitExternalSemaphoresAsync(
    hipExternalSemaphore_t const*         semsArray,
    hipExternalSemaphoreWaitParams const* paramsArray,
    unsigned int                           numExtSems) {
    HANDLE_ERROR(hipWaitExternalSemaphoresAsync(semsArray, paramsArray,
                                                 numExtSems, mStream));
}

void CUDAStream::SignalExternalSemaphoresAsyn(
    hipExternalSemaphore_t const*           semsArray,
    hipExternalSemaphoreSignalParams const* paramsArray,
    unsigned int                             numExtSems) {
    HANDLE_ERROR(hipSignalExternalSemaphoresAsync(semsArray, paramsArray,
                                                   numExtSems, mStream));
}

}  // namespace CUDA